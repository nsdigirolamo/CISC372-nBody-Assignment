#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "pvector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

vector3* host_velocities;
vector3* host_positions;
double* host_masses;

vector3* device_velocities;
vector3* device_positions;
double* device_masses;

vector3** dists;
vector3** accels;
vector3* accel_sums;

void initHostMemory (int numObjects) {

	host_velocities = (vector3*) malloc(sizeof(vector3) * numObjects);
	host_positions = (vector3*) malloc(sizeof(vector3) * numObjects);
	host_masses = (double*) malloc(sizeof(double) * numObjects);

}

// todo: make this a kernel so we can just make all that data local to the device only.

void initDeviceMemory (int numObjects) {

	// Allocating device memory for velocities, positions, masses, and acceleration sums

	hipMalloc(&device_velocities, sizeof(vector3) * numObjects);
	hipMalloc(&device_positions, sizeof(vector3) * numObjects);
	hipMalloc(&device_masses, sizeof(double) * numObjects);
	hipMalloc(&accel_sums, sizeof(vector3) * numObjects);

	// Allocating device memory for distances

	hipMalloc(&dists, sizeof(vector3*) * numObjects);
	vector3* host_dists[numObjects];
	for (int i = 0; i < numObjects; i++) {
		hipMalloc(&host_dists[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(dists, host_dists, sizeof(vector3*) * numObjects, hipMemcpyHostToDevice);

	// Allocating device memory for accelerations

	hipMalloc(&accels, sizeof(vector3*) * numObjects);
	vector3* host_accels[numObjects];
	for (int i = 0; i < numObjects; i++) {
		hipMalloc(&host_accels[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(accels, host_accels, sizeof(vector3*) * numObjects, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in initDeviceMemory! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif
}

void copyHostToDevice (int numObjects) {

	hipMemcpy(device_velocities, host_velocities, sizeof(vector3) * numObjects, hipMemcpyHostToDevice);
	hipMemcpy(device_positions, host_positions, sizeof(vector3) * numObjects, hipMemcpyHostToDevice);
	hipMemcpy(device_masses, host_masses, sizeof(double) * numObjects, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyHostToDevice! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif
}

void copyDeviceToHost (int numObjects) {

	hipMemcpy(host_velocities, device_velocities, sizeof(vector3) * numObjects, hipMemcpyDeviceToHost);
	hipMemcpy(host_positions, device_positions, sizeof(vector3) * numObjects, hipMemcpyDeviceToHost);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyDeviceToHost! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif
}

void freeHostMemory () {

	free(host_velocities);
	free(host_positions);
	free(host_masses);

}

void freeDeviceMemory () {

	hipFree(device_velocities);
	hipFree(device_positions);
	hipFree(device_masses);
	hipFree(dists);
	hipFree(accels);
	hipFree(accel_sums);

}

void planetFill () {

	int i, j;
	double data[][7] = {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE};

	for (i = 0; i <= NUMPLANETS;i ++) {
		for (j = 0; j < 3; j++) {
			host_positions[i][j] = data[i][j];
			host_velocities[i][j] = data[i][j+3];
		}
		host_masses[i]=data[i][6];
	}
}

void randomFill (int start, int count) {

	int i, j = start;

	for (i = start; i < start + count; i++) {
		for (j = 0; j < 3; j++) {
			host_velocities[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			host_positions[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			host_masses[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

void printSystem(FILE* handle){

	int i, j;

	for (i = 0; i < NUMENTITIES; i++) {

		fprintf(handle, "pos=(");
		for (j = 0; j < 3; j++) {
			fprintf(handle, "%lf,", host_positions[i][j]);
		}

		printf("),v=(");
		for (j = 0; j < 3; j++) {
			fprintf(handle, "%lf,", host_velocities[i][j]);
		}

		fprintf(handle,"),m=%lf\n",host_masses[i]);
	}
}

int main(int argc, char **argv)
{
	clock_t t0 = clock();
	int t_now;

	srand(1234);
	initHostMemory(NUMENTITIES);
	initDeviceMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	copyHostToDevice(NUMENTITIES);

	#ifdef DEBUG
	printSystem(stdout);
	#endif

	for (t_now=0;t_now<DURATION;t_now+=INTERVAL) {
		compute();
	}

	copyDeviceToHost(NUMENTITIES);

	clock_t t1 = clock() - t0;

	#ifdef DEBUG
	printSystem(stdout);
	#endif

	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
}
