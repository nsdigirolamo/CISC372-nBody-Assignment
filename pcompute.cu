#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

#define THREAD_MAXIMUM 1024

__global__ void calculateAccelerations (
		vector3* hVel, 
		vector3* hPos, 
		double* mass, 
		vector3* values, 
		vector3** accels, 
		int threads_per_block
	) {

	int row = blockIdx.x;
	int col = (threads_per_block * blockIdx.y) + threadIdx.x;

	if (NUMENTITIES <= col) return; // right now, the row variable will never be greater than NUMENTITIES

	if (row == col) { 

		FILL_VECTOR(accels[row][col], 0, 0, 0); 

	} else {

		vector3 distance;

		for (int i = 0; i < 3; i++) {
			distance[i] = hPos[row][i] - hPos[col][i];
		}

		/**
		 * Below's incredibly horrible line of code is brought to you by CUDA's implementation of fused multiply-add.
		 * Fused multiply-add is supposed to be faster and more accurate than separate operations, but it causes
		 * the math to differ from the CPU's math. So we need to do the below to disable it.
		 * 
		 * Here is where I found the solution to this: 
		 * https://stackoverflow.com/questions/14406364/different-results-for-cuda-addition-on-host-and-on-gpu
		 * And here is some more in-depth reading: 
		 * https://docs.nvidia.com/cuda/floating-point/index.html
		 */
		double magnitude_sq = __dadd_rn(__dadd_rn(__dmul_rn(distance[0], distance[0]), __dmul_rn(distance[1], distance[1])), __dmul_rn(distance[2], distance[2]));
		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * mass[col] / magnitude_sq;

		FILL_VECTOR(
			accels[row][col],
			accelmag * distance[0] / magnitude,
			accelmag * distance[1] / magnitude,
			accelmag * distance[2] / magnitude
		);

	}
}

void compute () {

	int i, j, k;

	vector3* values;
	vector3** accels;

	hipMallocManaged(&values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	hipMallocManaged(&accels, sizeof(vector3*) * NUMENTITIES);

	for (i = 0; i < NUMENTITIES; i++) {
		accels[i] = &values[i * NUMENTITIES];
	}

	int blocks_per_row = ceil((double)(NUMENTITIES) / (double)(THREAD_MAXIMUM));
	int threads_per_block = THREAD_MAXIMUM < NUMENTITIES ? (THREAD_MAXIMUM / 3) : NUMENTITIES;

	dim3 blocks(NUMENTITIES, blocks_per_row);
	dim3 threads(threads_per_block);

	calculateAccelerations<<<blocks, threads>>>(hVel, hPos, mass, values, accels, threads_per_block);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Kernel Launch Failed with Error: %s\n", hipGetErrorString(err));
	
	hipDeviceSynchronize();

	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]+=hVel[i][k]*INTERVAL;
		}
	}

	hipFree(accels);
	hipFree(values);
}
