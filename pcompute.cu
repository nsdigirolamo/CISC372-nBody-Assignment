#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

#define THREAD_MAXIMUM 1024
#define BLOCKS_PER_ROW (ceil((double)(NUMENTITIES) / (double)(THREAD_MAXIMUM)))
#define THREADS_PER_BLOCK (THREAD_MAXIMUM < NUMENTITIES ? THREAD_MAXIMUM : NUMENTITIES)

__global__ void calculateAccelerations (vector3** accels, vector3* hPos, double* masses) {

	int row = blockIdx.x;
	int col = (THREADS_PER_BLOCK * blockIdx.y) + threadIdx.x;

	if (NUMENTITIES <= col) return;

	if (row == col) {

		FILL_VECTOR(accels[row][col], 0, 0, 0);

	} else {

		vector3 distance;

		for (int i = 0; i < 3; i++) {
			distance[i] = hPos[row][i] - hPos[col][i];
		}

		#ifdef STRICT_CALC_ACCELS

		/**
		 * Below's line of code is brought to you by CUDA's fused multiply-add. Fused multiply-add is faster and 
		 * more accurate than separate operations, but it causes the math to differ from the CPU's math. 
		 * So we need to do the below to disable it. Also it doesn't seem like fused multiple-add is *that* much
		 * faster than doing it the regular way, so we should be fine keeping it like below.
		 * 
		 * Here is where I found the solution to this: 
		 * https://stackoverflow.com/questions/14406364/different-results-for-cuda-addition-on-host-and-on-gpu
		 * 
		 * And here is some more in-depth reading: 
		 * https://docs.nvidia.com/cuda/floating-point/index.html
		 * 
		 */

		double magnitude_sq = __dadd_rn(__dadd_rn(__dmul_rn(distance[0], distance[0]), __dmul_rn(distance[1], distance[1])), __dmul_rn(distance[2], distance[2]));

		#else

		double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];

		#endif

		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * masses[col] / magnitude_sq;

		FILL_VECTOR(
			accels[row][col],
			accelmag * distance[0] / magnitude,
			accelmag * distance[1] / magnitude,
			accelmag * distance[2] / magnitude
		);
	}
}

__global__ void sumAccelerations (vector3** accels, vector3* accel_sums) {

	int row = blockIdx.x;
	int col = (THREADS_PER_BLOCK * blockIdx.y) + threadIdx.x;

	if (col != 0) return;

	for (int i = 0; i < 3; i++) {
		accel_sums[row][i] = 0;
	}

	for (int i = 0; i < NUMENTITIES; i++) {
		for (int j = 0; j < 3; j++) {
			accel_sums[row][j] += accels[row][i][j];
		}
	}
}

void compute () {

	dim3 blocks (NUMENTITIES, BLOCKS_PER_ROW);
	dim3 threads (THREADS_PER_BLOCK);

	// Calculate Accelerations Starts Here

	vector3* accel_values;
	hipMallocManaged(&accel_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	vector3** accels;
	hipMallocManaged(&accels, sizeof(vector3*) * NUMENTITIES);

	for (int i = 0; i < NUMENTITIES; i++) {
		accels[i] = &accel_values[i * NUMENTITIES];
	}

	calculateAccelerations<<<blocks, threads>>>(accels, hPos, mass);
	hipError_t calculate_accelerations_error = hipGetLastError();
	if (calculate_accelerations_error != hipSuccess) 
		printf("calculateAccelerations kernel launch failed with Error: %s\n",
			hipGetErrorString(calculate_accelerations_error)
		);
	hipDeviceSynchronize();

	// Summing Accelerations Starts Here

	vector3* accel_sums;
	hipMallocManaged(&accel_sums, sizeof(vector3) * NUMENTITIES);

	sumAccelerations<<<blocks, threads>>>(accels, accel_sums);
	hipError_t sum_accelerations_error = hipGetLastError();
	if (sum_accelerations_error != hipSuccess) 
		printf("sumAccelerations kernel launch failed with Error: %s\n",
			hipGetErrorString(sum_accelerations_error)
		);
	hipDeviceSynchronize();

	// Calculating Changes Starts Here

	for (int i = 0; i < NUMENTITIES; i++) {
		for (int j = 0; j < 3; j++){
			hVel[i][j] += accel_sums[i][j] * INTERVAL;
			hPos[i][j] += hVel[i][j] * INTERVAL;
		}
	}

	hipFree(accel_values);
	hipFree(accels);
}
