#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

#define THREAD_MAXIMUM 1024

__global__ void calculateAccelerations (vector3* hVel, vector3* hPos, double* mass, vector3* values, vector3** accels, int threads_per_block) {

	int row = blockIdx.x;

	int first_col = threads_per_block * blockIdx.y;
	int col = first_col + threadIdx.x;

	if (NUMENTITIES <= col) {
		return;
	}

	if (row == col) { 

		FILL_VECTOR(accels[row][col], 0, 0, 0); 

	} else {

		vector3 distance;

		for (int i = 0; i < 3; i++) {
			distance[i] = hPos[row][i] - hPos[col][i];
		}

		double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * mass[col] / magnitude_sq;

		FILL_VECTOR(
			accels[row][col],
			accelmag * distance[0] / magnitude,
			accelmag * distance[1] / magnitude,
			accelmag * distance[2] / magnitude
		);

	}
}

void compute () {

	int i, j, k;

	vector3* values;
	vector3** accels;

	hipMallocManaged(&values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	hipMallocManaged(&accels, sizeof(vector3*) * NUMENTITIES);

	for (i = 0; i < NUMENTITIES; i++) {
		accels[i] = &values[i * NUMENTITIES];
	}

	int blocks_per_row = ceil((double)(NUMENTITIES) / (double)(THREAD_MAXIMUM));
	int threads_per_block = THREAD_MAXIMUM < NUMENTITIES ? (THREAD_MAXIMUM / 3) : NUMENTITIES;

	dim3 blocks(NUMENTITIES, blocks_per_row);
	dim3 threads(threads_per_block);

	calculateAccelerations<<<blocks, threads>>>(hVel, hPos, mass, values, accels, threads_per_block);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Kernel Launch Failed with Error: %s\n", hipGetErrorString(err));
	
	hipDeviceSynchronize();

	for (i=0;i<NUMENTITIES;i++){
		for (j=0;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}

	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]+=hVel[i][k]*INTERVAL;
		}
	}

	hipFree(accels);
	hipFree(values);
}
