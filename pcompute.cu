#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "pvector.h"
#include "config.h"

#define THREAD_MAXIMUM 1024
#define BLOCKS_PER_ROW (ceil((double)(NUMENTITIES) / (double)(THREAD_MAXIMUM)))
#define THREADS_PER_BLOCK (THREAD_MAXIMUM < NUMENTITIES ? THREAD_MAXIMUM : NUMENTITIES)
#define SPATIAL_AXES 3


__global__ void calcDists (vector3** dists, vector3* positions, double* masses) {

	int row = blockIdx.x;
	int col = (THREADS_PER_BLOCK * blockIdx.y) + threadIdx.x;
	int axis = blockIdx.z;

	if (NUMENTITIES <= col) return;

	if (row == col) {

		dists[row][col][axis] = 0;

	} else {

		dists[row][col][axis] = positions[row][axis] - positions[col][axis];

	}
}


__global__ void calcAccels (vector3** accels, vector3** dists, double* masses) {

	int row = blockIdx.x;
	int col = (THREADS_PER_BLOCK * blockIdx.y) + threadIdx.x;
	int axis = blockIdx.z;

	if (NUMENTITIES <= col) return;

	if (row == col) {

		accels[row][col][axis] = 0;

	} else {

		#ifdef STRICT_ACCELS

		double magnitude_sq = __dadd_rn(__dadd_rn(__dmul_rn(dists[row][col][0], dists[row][col][0]), __dmul_rn(dists[row][col][1], dists[row][col][1])), __dmul_rn(dists[row][col][2], dists[row][col][2]));

		#else

		double magnitude_sq = dists[row][col][0] * dists[row][col][0] + dists[row][col][1] * dists[row][col][1] + dists[row][col][2] * dists[row][col][2];

		#endif

		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * masses[col] / magnitude_sq;
		accels[row][col][axis] = accelmag * dists[row][col][axis] / magnitude;

	}
}

__global__ void sumAccels (vector3** accels, vector3* accel_sums) {

	int row = blockIdx.x;
	int col = (THREADS_PER_BLOCK * blockIdx.y) + threadIdx.x;
	int axis = blockIdx.z;

	if (col != 0) return;

	double accel_sum = 0;

	for (int i = 0; i < NUMENTITIES; i++) {
		accel_sum += accels[row][i][axis];
	}

	accel_sums[row][axis] = accel_sum;
}

__global__ void calcChanges (vector3* accel_sums, vector3* velocities, vector3* positions) {

	int row = blockIdx.x;
	int col = (THREADS_PER_BLOCK * blockIdx.y) + threadIdx.x;
	int axis = blockIdx.z;

	if (col != 0) return;

	velocities[row][axis] += accel_sums[row][axis] * INTERVAL;
	positions[row][axis] += velocities[row][axis] * INTERVAL;
}

void compute () {

	dim3 blocks (NUMENTITIES, BLOCKS_PER_ROW, SPATIAL_AXES);
	dim3 threads (THREADS_PER_BLOCK);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error before compute! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif

	// Calculate Distances
	calcDists<<<blocks, threads>>>(dists, device_positions, device_masses);
	#ifdef DEBUG
	hipError_t calc_dists_error = hipGetLastError();
	if (calc_dists_error != hipSuccess)
		printf("calcDists kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_dists_error),
			hipGetErrorString(calc_dists_error)
		);
	#endif
	hipDeviceSynchronize();

	// Calculate Accelerations
	calcAccels<<<blocks, threads>>>(accels, dists, device_masses);
	#ifdef DEBUG
	hipError_t calc_accels_error = hipGetLastError();
	if (calc_accels_error != hipSuccess) 
		printf("calcAccels kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_accels_error),
			hipGetErrorString(calc_accels_error)
		);
	#endif
	hipDeviceSynchronize();

	// Sum Accelerations
	sumAccels<<<blocks, threads>>>(accels, accel_sums);
	#ifdef DEBUG
	hipError_t sum_accels_error = hipGetLastError();
	if (sum_accels_error != hipSuccess) 
		printf("sumAccels kernel launch failed! %s: %s\n",
			hipGetErrorName(sum_accels_error),
			hipGetErrorString(sum_accels_error)
		);
	#endif
	hipDeviceSynchronize();

	// Calculating Changes
	calcChanges<<<blocks, threads>>>(accel_sums, device_velocities, device_positions);
	#ifdef DEBUG
	hipError_t calc_changes_error = hipGetLastError();
	if (calc_changes_error != hipSuccess) 
		printf("calcChanges kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_changes_error),
			hipGetErrorString(calc_changes_error)
		);
	#endif
	hipDeviceSynchronize();
}
