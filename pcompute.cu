#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

__global__ void do_print_test () {
	printf("Thread %d checking in from block %d!", blockIdx, threadIdx);
}

extern "C" // Required because nvcc treats .cu like .cpp code. This tells it to treat it like C code.
void compute () {

	/**
	 * num_blocks
	 * Can be int or dim3
	 * int - the number of blocks arranged in a 1D array
	 * dim3 - the number of blocks and their configuration in a grid 
	 */
	int num_blocks = 1;
	// Same limitations as above. Maximum 1024.
	int threads_per_block = 1;

	// Do the cuda thing.
	do_print_test<<<num_blocks, threads_per_block>>>();

	// Wait for completion.
	hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

	int i, j, k;

	vector3* values = (vector3*) malloc(sizeof(vector3) * NUMENTITIES * NUMENTITIES);
	vector3** accels = (vector3**) malloc(sizeof(vector3*) * NUMENTITIES);

	for (i = 0; i < NUMENTITIES; i++) {
		accels[i] = &values[i * NUMENTITIES];
	}

	for (i = 0; i < NUMENTITIES; i++) {
		for (j = 0; j < NUMENTITIES; j++) {
			if (i == j) {

				FILL_VECTOR(accels[i][j], 0, 0, 0); 

			} else {

				vector3 distance;

				for (k = 0; k < 3; k++) {
					distance[k] = hPos[i][k] - hPos[j][k];
				}

				double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
				double magnitude = sqrt(magnitude_sq);
				double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;

				FILL_VECTOR(
					accels[i][j],
					accelmag * distance[0] / magnitude,
					accelmag * distance[1] / magnitude,
					accelmag * distance[2] / magnitude
				);
			}
		}
	}

	for (i = 0; i < NUMENTITIES; i++) {

		vector3 accel_sum={0,0,0};

		for (j = 0; j < NUMENTITIES; j++) {
			for (k = 0; k < 3; k++) {
				accel_sum[k] += accels[i][j][k];
			}
		}

		for (k = 0; k < 3; k++){
			hVel[i][k] += accel_sum[k] * INTERVAL;
			hPos[i][k] += hVel[i][k] * INTERVAL;
		}
	}

	free(accels);
	free(values);
}
