#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

vector3* host_velocities;
vector3* host_positions;
double* host_masses;

int blocks_per_row;
int threads_per_block;

vector3* device_velocities;
vector3* device_positions;
double* device_masses;

vector3** dists;
vector3** accels;
vector3* accel_sums;

void initHostMemory (int numObjects) {

	host_velocities = (vector3*) malloc(sizeof(vector3) * numObjects);
	host_positions = (vector3*) malloc(sizeof(vector3) * numObjects);
	host_masses = (double*) malloc(sizeof(double) * numObjects);

}

void initBlocksAndThreads () {

	// Setting initial value for warp_groups_per_row

	int warp_groups_per_row = ceil((double)(NUMENTITIES) / (double)(WARP_GROUP_SIZE * THREADS_PER_WARP));

	// Everything below is to calculate how many blocks we have per row and how many threads we have per block.

	int warp_groups_exceed_max = MAX_WARP_GROUPS_PER_BLOCK < warp_groups_per_row;
	int leftover_warp_groups = warp_groups_per_row % MAX_WARP_GROUPS_PER_BLOCK;

	if (!warp_groups_exceed_max) {

		/**
		 * If we don't exceed the maximum number of warp groups, we have one block per row.
		 */

		blocks_per_row = 1;
		threads_per_block = warp_groups_per_row * THREADS_PER_WARP_GROUP;

	} else if (leftover_warp_groups == 0) {

		/**
		 * If we have more warp groups per row than our maximum but there are no leftover warp groups,
		 * that means that there is the maximum number of warp groups in all blocks.
		 */

		blocks_per_row = warp_groups_per_row / MAX_WARP_GROUPS_PER_BLOCK;
		threads_per_block = MAX_WARP_GROUPS_PER_BLOCK * THREADS_PER_WARP_GROUP;

	} else {

		/**
		 * If we're here, that means we have more warp groups per row than we can hold in a single
		 * block, and the number of warp groups per row doesn't divide equally across the blocks.
		 * We have leftoever warp groups that need to be put in a block, but all blocks are already
		 * maxxed out. So we need to create a new block and then evenly redistribute our warp groups
		 * into the blocks. 
		 */

		// This will give us one more block so we have room for our leftover warp groups.
		blocks_per_row = ceil((double)(warp_groups_per_row) / (double)(MAX_WARP_GROUPS_PER_BLOCK));
		// This will give us the number of warp groups per block.
		int warp_groups_per_block = ceil((double)(warp_groups_per_row) / (double)(blocks_per_row));
		// Now we can get the number of threads per block by multiplying.
		threads_per_block = warp_groups_per_block * THREADS_PER_WARP_GROUP;

		#ifdef BLOCKS_THREADS_INIT_INFO
		warp_groups_per_row = blocks_per_row * warp_groups_per_block;
		#endif
	}

	#ifdef BLOCKS_THREADS_INIT_INFO
	printf("warp_groups_per_row %d\nblocks_per_row %d\nthreads_per_block %d\n",
		warp_groups_per_row,
		blocks_per_row,
		threads_per_block
	);
	#endif
}

// todo: make this a kernel so we can just make all that data local to the device only.

void initDeviceMemory (int numObjects) {

	// Allocating device memory for velocities, positions, masses, and acceleration sums

	hipMalloc(&device_velocities, sizeof(vector3) * numObjects);
	hipMalloc(&device_positions, sizeof(vector3) * numObjects);
	hipMalloc(&device_masses, sizeof(double) * numObjects);
	hipMalloc(&accel_sums, sizeof(vector3) * numObjects);

	// Allocating device memory for distances

	hipMalloc(&dists, sizeof(vector3*) * numObjects);
	vector3* host_dists[numObjects];
	for (int i = 0; i < numObjects; i++) {
		hipMalloc(&host_dists[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(dists, host_dists, sizeof(vector3*) * numObjects, hipMemcpyHostToDevice);

	// Allocating device memory for accelerations

	hipMalloc(&accels, sizeof(vector3*) * numObjects);
	vector3* host_accels[numObjects];
	for (int i = 0; i < numObjects; i++) {
		hipMalloc(&host_accels[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(accels, host_accels, sizeof(vector3*) * numObjects, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in initDeviceMemory! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif
}

void copyHostToDevice (int numObjects) {

	hipMemcpy(device_velocities, host_velocities, sizeof(vector3) * numObjects, hipMemcpyHostToDevice);
	hipMemcpy(device_positions, host_positions, sizeof(vector3) * numObjects, hipMemcpyHostToDevice);
	hipMemcpy(device_masses, host_masses, sizeof(double) * numObjects, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyHostToDevice! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif
}

void copyDeviceToHost (int numObjects) {

	hipMemcpy(host_velocities, device_velocities, sizeof(vector3) * numObjects, hipMemcpyDeviceToHost);
	hipMemcpy(host_positions, device_positions, sizeof(vector3) * numObjects, hipMemcpyDeviceToHost);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyDeviceToHost! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	#endif
}

void freeHostMemory () {

	free(host_velocities);
	free(host_positions);
	free(host_masses);

}

void freeDeviceMemory () {

	hipFree(device_velocities);
	hipFree(device_positions);
	hipFree(device_masses);

	/**
	 * TODO: I don't think dists is getting freed properly. Need the free the rows
	 * before freeing the double pointer?
	 */

	hipFree(dists);
	hipFree(accels);
	hipFree(accel_sums);

}

void planetFill () {

	int i, j;
	double data[][7] = {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE};

	for (i = 0; i <= NUMPLANETS;i ++) {
		for (j = 0; j < 3; j++) {
			host_positions[i][j] = data[i][j];
			host_velocities[i][j] = data[i][j+3];
		}
		host_masses[i]=data[i][6];
	}
}

void randomFill (int start, int count) {

	int i, j = start;

	for (i = start; i < start + count; i++) {
		for (j = 0; j < 3; j++) {
			host_velocities[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			host_positions[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			host_masses[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

void printSystem(FILE* handle){

	int i, j;

	for (i = 0; i < NUMENTITIES; i++) {

		fprintf(handle, "pos=(");
		for (j = 0; j < 3; j++) {
			fprintf(handle, "%lf,", host_positions[i][j]);
		}

		printf("),v=(");
		for (j = 0; j < 3; j++) {
			fprintf(handle, "%lf,", host_velocities[i][j]);
		}

		fprintf(handle,"),m=%lf\n",host_masses[i]);
	}
}

int main(int argc, char **argv)
{
	clock_t t0 = clock();
	int t_now;

	srand(1234);
	initHostMemory(NUMENTITIES);
	initDeviceMemory(NUMENTITIES);
	initBlocksAndThreads();
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	copyHostToDevice(NUMENTITIES);

	#ifdef DEBUG
	printSystem(stdout);
	#endif

	for (t_now=0; t_now < DURATION; t_now += INTERVAL) {
		compute();
	}

	copyDeviceToHost(NUMENTITIES);

	clock_t t1 = clock() - t0;

	#ifdef DEBUG
	printSystem(stdout);
	#endif

	printf("This took a total time of %f seconds\n", (double)(t1) / CLOCKS_PER_SEC);

	freeHostMemory();
	freeDeviceMemory();
}
