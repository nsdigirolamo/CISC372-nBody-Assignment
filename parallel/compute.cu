#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

int calcGridDim (int block_width, int entity_count) {

	if (entity_count < block_width) return 1;

	int grid_width = entity_count / block_width;

	if (entity_count % block_width == 0) return grid_width;

	return grid_width + 1;
}

__global__ void calcAccels (vector3** accels, vector3* positions, double* masses) {

	int local_row = threadIdx.y;
	int local_col = threadIdx.x;

	int global_row = (blockIdx.y * blockDim.y) + local_row;
	int global_col = (blockIdx.x * blockDim.x) + local_col;
	int spatial_axis = threadIdx.z;

	if (NUMENTITIES <= global_col || NUMENTITIES <= global_row) return;

	if (global_row == global_col) {

		accels[global_row][global_col][spatial_axis] = 0;

	} else {

		__shared__ vector3 distances[CALC_ACCELS_BLOCK_WIDTH][CALC_ACCELS_BLOCK_WIDTH];

		distances[local_row][local_col][spatial_axis] = positions[global_row][spatial_axis] - positions[global_col][spatial_axis];

		__syncthreads();

		double magnitude_sq = distances[local_row][local_col][0] * distances[local_row][local_col][0] + distances[local_row][local_col][1] * distances[local_row][local_col][1] + distances[local_row][local_col][2] * distances[local_row][local_col][2];
		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * masses[global_col] / magnitude_sq;
		accels[global_row][global_col][spatial_axis] = accelmag * distances[local_row][local_col][spatial_axis] / magnitude;

	}
}

__global__ void sumAccels (vector3** accels, int global_sum_length) {

	/**
	 * I used this resource to help me optimize my code.
	 * https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
	 */

	int local_col = threadIdx.x;

	int global_row = blockIdx.y;
	// Double our offsets because each thread is assigned two data points globally
	int global_col = (blockIdx.x * blockDim.x * 2) + (local_col * 2);
	int spatial_axis = threadIdx.z;

	__shared__ vector3 sums[SUM_LENGTH];

	/**
	 * We need to zero out the values in sums because some local_cols may not be
	 * needed. If they don't zero out their values, some other thread may try
	 * to add a garbage value to their own good value.
	 */
	sums[local_col][spatial_axis] = 0;

	if (global_sum_length <= global_col) return;

	// Below initializes the shared array with summed global values.

	int offset = 1;
	bool neighbor_exceeds_bounds = global_sum_length <= global_col + offset;

	if (!neighbor_exceeds_bounds) {

		sums[local_col][spatial_axis] = accels[global_row][global_col][spatial_axis] + accels[global_row][global_col + 1][spatial_axis];

	} else {

		sums[local_col][spatial_axis] = accels[global_row][global_col][spatial_axis];

	}

	// We will now treat the local_col as a thread id
	int thread_id = local_col;

	for (offset = 1; offset < SUM_LENGTH; offset *= 2) {
		// This will produce a strided index from the thread_id
		int i = 2 * offset * thread_id;
		// This allows for a non-divergent branch within the loop.
		if (i < SUM_LENGTH) {
			sums[i][spatial_axis] += sums[i + offset][spatial_axis];
		}
		__syncthreads();
	}

	if (local_col == 0) accels[global_row][blockIdx.x][spatial_axis] = sums[local_col][spatial_axis];
}

__global__ void calcChanges (vector3** accels, vector3* velocities, vector3* positions) {

	int local_row = threadIdx.y;
	int global_row = (blockIdx.y * blockDim.y) + local_row;
	int spatial_axis = blockIdx.z;

	if (NUMENTITIES <= global_row) return;

	velocities[global_row][spatial_axis] += accels[global_row][0][spatial_axis] * INTERVAL;
	positions[global_row][spatial_axis] += velocities[global_row][spatial_axis] * INTERVAL; 
}

void compute () {

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error before compute! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif

	// Calculate Accelerations

	calcAccels<<<calc_accels_grid_dims, calc_accels_block_dims>>>(accels, device_positions, device_masses);

	#ifdef DEBUG
	hipError_t calc_accels_error = hipGetLastError();
	if (calc_accels_error != hipSuccess) {
		printf("calcAccels kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_accels_error),
			hipGetErrorString(calc_accels_error)
		);
		printf("\tcalcAccels Config: gridDims: {%d %d %d}, blockDims: {%d %d %d}\n",
			calc_accels_grid_dims.x,
			calc_accels_grid_dims.y,
			calc_accels_grid_dims.z,
			calc_accels_block_dims.x,
			calc_accels_block_dims.y,
			calc_accels_block_dims.z
		);
	}
	fflush(stdout);
	#endif

	// Sum Accelerations

	int global_sum_length = NUMENTITIES;
	dim3 sum_block_dims (SUM_LENGTH, 1, SPATIAL_DIMS);

	while (1 < global_sum_length) {

		int sum_grid_width = calcGridDim(SUM_LENGTH * 2, global_sum_length); // Multiply by two because each thread reduces two data points in accels.
		dim3 sum_grid_dims (sum_grid_width, NUMENTITIES, 1);

		sumAccels<<<sum_grid_dims, sum_block_dims>>>(accels, global_sum_length);

		#ifdef DEBUG
		hipError_t sum_accels_error = hipGetLastError();
		if (sum_accels_error != hipSuccess) {
			printf("sumAccels kernel launch failed! %s: %s\n",
				hipGetErrorName(sum_accels_error),
				hipGetErrorString(sum_accels_error)
			);
			printf("\tsumAccels Config: gridDims: {%d %d %d}, blockDims: {%d %d %d}\n",
				sum_grid_dims.x,
				sum_grid_dims.y,
				sum_grid_dims.z,
				sum_block_dims.x,
				sum_block_dims.y,
				sum_block_dims.z
			);
		}
		fflush(stdout);
		#endif

		global_sum_length = sum_grid_width;
	}

	// Calculating Changes

	calcChanges<<<calc_changes_grid_dims, calc_changes_block_dims>>>(accels, device_velocities, device_positions);

	#ifdef DEBUG
	hipError_t calc_changes_error = hipGetLastError();
	if (calc_changes_error != hipSuccess) {
		printf("calcChanges kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_changes_error),
			hipGetErrorString(calc_changes_error)
		);
		printf("\tcalcChanges Config: gridDims: {%d %d %d}, blockDims: {%d %d %d}\n",
			calc_changes_grid_dims.x,
			calc_changes_grid_dims.y,
			calc_changes_grid_dims.z,
			calc_changes_block_dims.x,
			calc_changes_block_dims.y,
			calc_changes_block_dims.z
		);
	}
	fflush(stdout);
	#endif
}
