#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

int calcGridDim (int block_width, int entity_count) {

	if (entity_count < block_width) return 1;

	int grid_width = entity_count / block_width;

	if (entity_count % block_width == 0) return grid_width;

	return grid_width + 1;
}

__global__ void calcAccels (vector3** accels, vector3* positions, double* masses) {

	int local_row = threadIdx.y;
	int local_col = threadIdx.x;

	int global_row = (blockIdx.y * blockDim.y) + local_row;
	int global_col = (blockIdx.x * blockDim.x) + local_col;
	int spatial_axis = threadIdx.z;

	if (NUMENTITIES <= global_col || NUMENTITIES <= global_row) return;

	if (global_row == global_col) {

		accels[global_row][global_col][spatial_axis] = 0;

	} else {

		__shared__ vector3 distances[SQUARE_SIZE][SQUARE_SIZE];

		distances[local_row][local_col][spatial_axis] = positions[global_row][spatial_axis] - positions[global_col][spatial_axis];

		__syncthreads();

		double magnitude_sq = distances[local_row][local_col][0] * distances[local_row][local_col][0] + distances[local_row][local_col][1] * distances[local_row][local_col][1] + distances[local_row][local_col][2] * distances[local_row][local_col][2];
		double magnitude = sqrt(magnitude_sq);
		double accelmag = -1 * GRAV_CONSTANT * masses[global_col] / magnitude_sq;
		accels[global_row][global_col][spatial_axis] = accelmag * distances[local_row][local_col][spatial_axis] / magnitude;

	}
}

__global__ void sumAccels (vector3** accels) {

	int global_row = blockIdx.y;
	int spatial_axis = threadIdx.z;
	
	double sum = 0;

	for (int i = 0; i < NUMENTITIES; i++) {
		sum += accels[global_row][i][spatial_axis];
	}

	accels[global_row][0][spatial_axis] = sum;
}

__global__ void calcChanges (vector3** accels, vector3* velocities, vector3* positions) {

	int local_row = threadIdx.y;
	int global_row = (blockIdx.y * blockDim.y) + local_row;
	int spatial_axis = threadIdx.z;

	if (NUMENTITIES <= global_row) return;

	velocities[global_row][spatial_axis] += accels[global_row][0][spatial_axis] * INTERVAL;
	positions[global_row][spatial_axis] += velocities[global_row][spatial_axis] * INTERVAL; 
}

void compute () {

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error before compute! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif

	// Calculate Accelerations

	int accels_grid_width = calcGridDim(SQUARE_SIZE, NUMENTITIES);
	dim3 accels_grid_dims (accels_grid_width, accels_grid_width, 1);
	dim3 accels_block_dims (SQUARE_SIZE, SQUARE_SIZE, SPATIAL_DIMS);

	calcAccels<<<accels_grid_dims, accels_block_dims>>>(accels, device_positions, device_masses);

	#ifdef DEBUG
	hipError_t calc_accels_error = hipDeviceSynchronize();
	if (calc_accels_error != hipSuccess) {
		printf("calcAccels kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_accels_error),
			hipGetErrorString(calc_accels_error)
		);
		printf("\tcalcAccels Config: gridDims: {%d %d %d}, blockDims: {%d %d %d}\n",
			accels_grid_dims.x,
			accels_grid_dims.y,
			accels_grid_dims.z,
			accels_block_dims.x,
			accels_block_dims.y,
			accels_block_dims.z
		);
	}
	fflush(stdout);
	#endif

	// Sum Accelerations

	/**
	 * TODO: This isn't the best right now. Only assigns three threads per row.
	 */
	dim3 sum_grid_dims (1, NUMENTITIES, 1);
	dim3 sum_block_dims (1, 1, SPATIAL_DIMS);

	sumAccels<<<sum_grid_dims, sum_block_dims>>>(accels);

	#ifdef DEBUG
	hipError_t sum_accels_error = hipDeviceSynchronize();;
	if (sum_accels_error != hipSuccess) {
		printf("sumAccels kernel launch failed! %s: %s\n",
			hipGetErrorName(sum_accels_error),
			hipGetErrorString(sum_accels_error)
		);
		printf("\tsumAccels Config: gridDims: {%d %d %d}, blockDims: {%d %d %d}\n",
			sum_grid_dims.x,
			sum_grid_dims.y,
			sum_grid_dims.z,
			sum_block_dims.x,
			sum_block_dims.y,
			sum_block_dims.z
		);
	}
	fflush(stdout);
	#endif

	// Calculating Changes

	calcChanges<<<calc_changes_grid_dims, calc_changes_block_dims>>>(accels, device_velocities, device_positions);

	#ifdef DEBUG
	hipError_t calc_changes_error = hipDeviceSynchronize();;
	if (calc_changes_error != hipSuccess) {
		printf("calcChanges kernel launch failed! %s: %s\n",
			hipGetErrorName(calc_changes_error),
			hipGetErrorString(calc_changes_error)
		);
		printf("\tcalcChanges Config: gridDims: {%d %d %d}, blockDims: {%d %d %d}\n",
			calc_changes_grid_dims.x,
			calc_changes_grid_dims.y,
			calc_changes_grid_dims.z,
			calc_changes_block_dims.x,
			calc_changes_block_dims.y,
			calc_changes_block_dims.z
		);
	}
	fflush(stdout);
	#endif
}
