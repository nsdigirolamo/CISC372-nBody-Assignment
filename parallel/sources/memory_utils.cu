#include <stdlib.h>
#include <stdio.h>

#include "config.cuh"
#include "nbody.cuh"
#include "vector.cuh"

// Host Memory

vector3* host_velocities;
vector3* host_positions;
double* host_masses;

// Device Memory

vector3* device_velocities;
vector3* device_positions;
double* device_masses;

size_t accels_pitch;
vector3* accels;

/**
 * Initializes memory on the host.
 */
void initHostMemory () {

	host_velocities = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_positions = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_masses = (double*) malloc(sizeof(double) * NUMENTITIES);

}

/**
 * Initializes memory on the device.
 */
void initDeviceMemory () {

	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory filter");
	#endif

	hipMalloc(&device_velocities, sizeof(vector3) * NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory velocities");
	#endif

	hipMalloc(&device_positions, sizeof(vector3) * NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory positions");
	#endif

	hipMalloc(&device_masses, sizeof(double) * NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory masses");
	#endif

	hipMallocPitch(&accels, &accels_pitch, sizeof(vector3) * NUMENTITIES, NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory accels");
	#endif
}

/**
 * Copies memory from the host to the device.
 */
void copyHostToDevice () {

	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice filter");
	#endif

	hipMemcpy(device_velocities, host_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice velocities");
	#endif

	hipMemcpy(device_positions, host_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice positions");
	#endif

	hipMemcpy(device_masses, host_masses, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice masses");
	#endif
}

/**
 * Copies memory from the device to the host.
 */
void copyDeviceToHost () {

	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyDeviceToHost filter");
	#endif

	hipMemcpy(host_velocities, device_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "copyDeviceToHost velocities"); 
	#endif

	hipMemcpy(host_positions, device_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyDeviceToHost positions");
	#endif
}

/**
 * Frees the host memory.
 */
void freeHostMemory () {

	free(host_velocities);
	free(host_positions);
	free(host_masses);

}

/**
 * Frees the device memory.
 */
void freeDeviceMemory () {

	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "freeDeviceMemory filter");
	#endif

	hipFree(device_velocities);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory velocities"); 
	#endif

	hipFree(device_positions);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory positions"); 
	#endif

	hipFree(device_masses);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory masses"); 
	#endif

	hipFree(accels);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory accels"); 
	#endif
}