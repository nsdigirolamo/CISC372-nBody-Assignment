#include <stdlib.h>
#include <stdio.h>

#include "config.cuh"
#include "nbody.cuh"
#include "vector.cuh"

// Host Memory

vector3* host_velocities;
vector3* host_positions;
double* host_masses;

// Device Memory

vector3* device_velocities;
vector3* device_positions;
double* device_masses;

size_t accels_pitch;
vector3* accels;

void initHostMemory () {

	host_velocities = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_positions = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_masses = (double*) malloc(sizeof(double) * NUMENTITIES);

}

void initDeviceMemory () {\

	hipMalloc(&device_velocities, sizeof(vector3) * NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory velocities");
	#endif

	hipMalloc(&device_positions, sizeof(vector3) * NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory positions");
	#endif

	hipMalloc(&device_masses, sizeof(double) * NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory masses");
	#endif

	hipMallocPitch(&accels, &accels_pitch, sizeof(vector3) * NUMENTITIES, NUMENTITIES);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "initDeviceMemory accels");
	#endif
}

void copyHostToDevice () {

	hipMemcpy(device_velocities, host_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice velocities");
	#endif

	hipMemcpy(device_positions, host_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice positions");
	#endif

	hipMemcpy(device_masses, host_masses, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyHostToDevice masses");
	#endif
}

void copyDeviceToHost () {

	hipMemcpy(host_velocities, device_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "copyDeviceToHost velocities"); 
	#endif

	hipMemcpy(host_positions, device_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	#ifdef DEBUG
	handleCudaError(hipGetLastError(), "copyDeviceToHost positions");
	#endif
}

void freeHostMemory () {

	free(host_velocities);
	free(host_positions);
	free(host_masses);

}

void freeDeviceMemory () {

	hipFree(device_velocities);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory velocities"); 
	#endif

	hipFree(device_positions);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory positions"); 
	#endif

	hipFree(device_masses);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory masses"); 
	#endif

	hipFree(accels);
	#ifdef DEBUG 
	handleCudaError(hipGetLastError(), "freeDeviceMemory accels"); 
	#endif
}