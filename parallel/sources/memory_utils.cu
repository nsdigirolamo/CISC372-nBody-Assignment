#include "config.cuh"
#include "vector.cuh"

// Host Memory

vector3* host_velocities;
vector3* host_positions;
double* host_masses;

// Device Memory

vector3* device_velocities;
vector3* device_positions;
double* device_masses;

vector3** accels;

void initHostMemory () {

	host_velocities = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_positions = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_masses = (double*) malloc(sizeof(double) * NUMENTITIES);

}

void initDeviceMemory () {

	// Allocating device memory for velocities, positions, masses, and acceleration sums

	hipMalloc(&device_velocities, sizeof(vector3) * NUMENTITIES);
	hipMalloc(&device_positions, sizeof(vector3) * NUMENTITIES);
	hipMalloc(&device_masses, sizeof(double) * NUMENTITIES);

	// Allocating device memory for accelerations

	hipMalloc(&accels, sizeof(vector3*) * NUMENTITIES);
	vector3* host_accels[NUMENTITIES];
	for (int i = 0; i < NUMENTITIES; i++) {
		hipMalloc(&host_accels[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(accels, host_accels, sizeof(vector3*) * NUMENTITIES, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in initDeviceMemory! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif
}

void copyHostToDevice () {

	hipMemcpy(device_velocities, host_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(device_positions, host_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(device_masses, host_masses, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyHostToDevice! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif
}

void copyDeviceToHost () {

	hipMemcpy(host_velocities, device_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(host_positions, device_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyDeviceToHost! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif
}

void freeHostMemory () {

	free(host_velocities);
	free(host_positions);
	free(host_masses);

}

void freeDeviceMemory () {

	hipFree(device_velocities);
	hipFree(device_positions);
	hipFree(device_masses);

	/**
	 * TODO: I don't think this is freeing accels properly.
	 * Don't we have to free all the pointers in accels first,
	 * and then free accels itself?
	 */

	hipFree(accels);

}