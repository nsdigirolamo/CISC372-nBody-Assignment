#include <stdlib.h>
#include <stdio.h>

#include "config.cuh"
#include "vector.cuh"

// Host Memory

vector3* host_velocities;
vector3* host_positions;
double* host_masses;

// Device Memory

vector3* device_velocities;
vector3* device_positions;
double* device_masses;

size_t accels_pitch;
vector3* accels;

void initHostMemory () {

	host_velocities = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_positions = (vector3*) malloc(sizeof(vector3) * NUMENTITIES);
	host_masses = (double*) malloc(sizeof(double) * NUMENTITIES);

}

void initDeviceMemory () {

	hipMalloc(&device_velocities, sizeof(vector3) * NUMENTITIES);
	hipMalloc(&device_positions, sizeof(vector3) * NUMENTITIES);
	hipMalloc(&device_masses, sizeof(double) * NUMENTITIES);
	hipMallocPitch(&accels, &accels_pitch, sizeof(vector3) * (NUMENTITIES + 1), (NUMENTITIES + 1));

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in initDeviceMemory! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif
}

void copyHostToDevice () {

	hipMemcpy(device_velocities, host_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(device_positions, host_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(device_masses, host_masses, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyHostToDevice! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif
}

void copyDeviceToHost () {

	hipMemcpy(host_velocities, device_velocities, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(host_positions, device_positions, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);

	#ifdef DEBUG
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
		printf("Error in copyDeviceToHost! %s: %s\n",
			hipGetErrorName(e),
			hipGetErrorString(e)
		);
	fflush(stdout);
	#endif
}

void freeHostMemory () {

	free(host_velocities);
	free(host_positions);
	free(host_masses);

}

void freeDeviceMemory () {

	hipFree(device_velocities);
	hipFree(device_positions);
	hipFree(device_masses);
	hipFree(accels);

}